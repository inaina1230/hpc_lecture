
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__device__ __managed__ int sum;

__global__ void thread(int *a) { 
  a[threadIdx.x] = 0; 
}

__global__ void reduction(int *bucket,int *key) {
  int i = threadIdx.x;
  atomicAdd(&bucket[key[i]], 1);
}

__global__ void sort(int num,int *key,int sum) {
  int thread = threadIdx.x;
  key[sum+thread] = num;
}

int main() {
  int n = 50;
  int range = 5;
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  int *bucket;
  hipMallocManaged(&bucket, range*sizeof(int));
  thread<<<1,range>>>(bucket);
  reduction<<<1,n>>>(bucket,key);
  sum=0;
  for(int i=0;i<range;i++){
    int threadnum = bucket[i];
    sort<<<1,threadnum>>>(i,key,sum);
    sum += threadnum;
  }
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
}
